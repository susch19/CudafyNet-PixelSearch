
#include <hip/hip_runtime.h>
struct ProgramGPUColorRGB
{
	__device__  ProgramGPUColorRGB()
	{
	}
	unsigned char Blue;
	unsigned char Green;
	unsigned char Red;
	unsigned char Alpha;
};


// Insaniquarium_Deluxe_Bot.Program
extern "C" __global__  void FindPixel( ProgramGPUColorRGB* rgbColors, int rgbColorsLen0,  ProgramGPUColorRGB* colors, int colorsLen0,  int* indices, int indicesLen0,  float* output, int outputLen0);

// Insaniquarium_Deluxe_Bot.Program
extern "C" __global__  void FindPixel( ProgramGPUColorRGB* rgbColors, int rgbColorsLen0,  ProgramGPUColorRGB* colors, int colorsLen0,  int* indices, int indicesLen0,  float* output, int outputLen0)
{
	__syncthreads();
	indices[(threadIdx.x)]++;
}
